//
// Forward and backwards cuFFT Benchmar using graphs and callbacks.
//
// This test case measures the runtime of a set of function calls with the following order:
//   -> work on the real array (init kernel) 
//   -> forward FFT 
//   -> work on the complex array (work kernel)
//   -> backwards FFT 
//   -> work on the real array (post kernel)
//
// This benchmark compares the execution of the calls above using different methods
// available in CUDA. These are:
//   -> Regular independent calls on the same stream.
//   -> Packing all the calls into a CUDA graph.
//   -> Fusing the non-FFT kernels into the cuFFT calls with callbacks.
//   -> Packing the previous method into a graph.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

// This array layout is equivalent to a 2D array with the transforms done in the innermost dimension.
#define INPUT_SIGNAL_SIZE 128
#define BATCH_SIZE 20000
#define COMPLEX_SIGNAL_SIZE (INPUT_SIGNAL_SIZE/2 + 1)

#define BLOCK_SIZE 128
#define REPEAT 1000

// Regular kernels
__global__ void init(double * arr, int n)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < n) 
  {
    arr[id] = cos( ((double)id) / ((double)INPUT_SIGNAL_SIZE) * 10.0f * 3.141592f );
  }
}

__global__ void work (hipfftDoubleComplex * arr, int n)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if ( id < n ) 
  {
    arr[id] = hipfftDoubleComplex{2.0,2.0};
  }
}

__global__ void post(double * arr, int n)
{
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < n) 
  {
    arr[id] = arr[id] * arr[id] - 0.5f;
  }
}

// Device callbacks
__device__ double CB_init(void * dataIn, size_t offset, void * callerInfo, void * sharedPtr)
{
  return cos( ((double)offset) / ((double)INPUT_SIGNAL_SIZE) * 10.0f * 3.141592f );
}

__device__ void CB_work(void * dataOut, size_t offset, hipfftDoubleComplex element, void * callerInfo, void * sharedPtr)
{
  ((hipfftDoubleComplex*)dataOut)[offset] = hipfftDoubleComplex{2.0,2.0};
}

__device__ void CB_post(void * dataOut, size_t offset, hipfftDoubleReal element, void * callerInfo, void * sharedPtr)
{
  ((hipfftDoubleReal*)dataOut)[offset] = element * element - 0.5;
}

__managed__ hipfftCallbackLoadD d_loadCallbackInit = CB_init;
__managed__ hipfftCallbackStoreZ d_storeCallbackWork = CB_work;
__managed__ hipfftCallbackStoreD d_storeCallbackPost = CB_post;

int main()
{

  hipfftDoubleReal * in_d;
  hipfftDoubleComplex * out_d; 
 
  hipMalloc((void**)&in_d,  sizeof(hipfftDoubleReal)*INPUT_SIGNAL_SIZE*BATCH_SIZE);
  hipMalloc((void**)&out_d, sizeof(hipfftDoubleComplex)*COMPLEX_SIGNAL_SIZE*BATCH_SIZE);
 
  hipfftHandle planForward, planBackwards;
  int transformSizeIn = INPUT_SIGNAL_SIZE;
  int transformSizeOut = COMPLEX_SIGNAL_SIZE;
  hipfftPlanMany(&planForward, 1, &transformSizeIn, &transformSizeIn, 1, transformSizeIn, 
                                                   &transformSizeOut, 1, transformSizeOut,
                                                   HIPFFT_D2Z, BATCH_SIZE);
  hipfftPlanMany(&planBackwards, 1, &transformSizeOut, &transformSizeOut, 1, transformSizeOut, 
                                                      &transformSizeIn, 1, transformSizeIn,
                                                      HIPFFT_Z2D, BATCH_SIZE);
  // Attach plans to a cuda stream.
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipfftSetStream(planForward, stream);
  hipfftSetStream(planBackwards, stream);

  // Declare event variables for timing purposes
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*
    Regular independent call of the kernels and cufft functions.
  */
  hipEventRecord(start, 0);
  for ( int i = 0; i < REPEAT; i++)
  {
    // Initialize the input data.
    init<<<(INPUT_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(in_d, INPUT_SIGNAL_SIZE);
 
    // Forwards & backwards FFTs 
    hipfftExecD2Z(planForward, in_d, out_d); 
    work<<<(COMPLEX_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(out_d, COMPLEX_SIGNAL_SIZE);
    hipfftExecZ2D(planBackwards, out_d, in_d); 
    post<<<(INPUT_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(in_d, INPUT_SIGNAL_SIZE);
    hipStreamSynchronize(stream);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime, baseline;
  hipEventElapsedTime(&elapsedTime, start, stop);
  baseline = elapsedTime/(float)REPEAT;
  std::cout << "Elapsed time of the regular cufft was " << baseline << " ms" << std::endl;

  /* 
    Graph-based FFT launches
  */
  hipGraph_t graph;
  hipGraphExec_t instance;

  // Capure the graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  // Same work as with the loop above.
  init<<<(INPUT_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(in_d, INPUT_SIGNAL_SIZE);
  hipfftExecD2Z(planForward, in_d, out_d); 
  work<<<(COMPLEX_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(out_d, COMPLEX_SIGNAL_SIZE);
  hipfftExecZ2D(planBackwards, out_d, in_d); 
  post<<<(INPUT_SIGNAL_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE, 0, stream>>>(in_d, INPUT_SIGNAL_SIZE);
  
  // End stream Capture & instantiate the graph
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  // Repeat the work done above, but using graaphs this time
  hipEventRecord(start, 0);
  for ( int i = 0; i < REPEAT; i++)
  {
    hipGraphLaunch(instance, stream); 
    hipStreamSynchronize(stream);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Speed-up from using CUDA graphs: " << baseline/(elapsedTime/(float)REPEAT) << "X" << std::endl;

  /* 
    Same as above, but with callbacks
  */
  hipfftXtSetCallback(planForward, (void**)&d_loadCallbackInit, HIPFFT_CB_LD_REAL_DOUBLE, 0); 
  hipfftXtSetCallback(planForward, (void**)&d_storeCallbackWork, HIPFFT_CB_ST_COMPLEX_DOUBLE, 0); 
  hipfftXtSetCallback(planBackwards, (void**)&d_storeCallbackPost, HIPFFT_CB_ST_REAL_DOUBLE, 0); 

  hipEventRecord(start, 0);
  for ( int i = 0; i < REPEAT; i++)
  {
    // Forwards & backwards FFTs 
    hipfftExecD2Z(planForward, in_d, out_d); 
    hipfftExecZ2D(planBackwards, out_d, in_d); 
    hipStreamSynchronize(stream);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Speed-up form using cuFFT callbacks: " << baseline/(elapsedTime/(float)REPEAT) << "X" << std::endl;

  /* 
    Graph-based FFTs with callbacks 
  */
  hipGraph_t graph2;
  hipGraphExec_t instance2;

  // Capure the graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  // Same work as with the loop above.
  hipfftExecD2Z(planForward, in_d, out_d); 
  hipfftExecZ2D(planBackwards, out_d, in_d); 
  
  // End stream Capture & instantiate the graph
  hipStreamEndCapture(stream, &graph2);
  hipGraphInstantiate(&instance2, graph2, NULL, NULL, 0);

  // Repeat the work done above, but using graaphs this time
  hipEventRecord(start, 0);
  for ( int i = 0; i < REPEAT; i++)
  {
    hipGraphLaunch(instance2, stream); 
    hipStreamSynchronize(stream);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  std::cout << "Speed-up from using cuFFT callbacks and CUDA graphs: " << baseline/(elapsedTime/(float)REPEAT) << "X" << std::endl;

  // Free the memory
  hipfftDestroy(planForward);
  hipfftDestroy(planBackwards);
  hipFree(in_d); 
  hipFree(out_d); 

  return 0;

}
